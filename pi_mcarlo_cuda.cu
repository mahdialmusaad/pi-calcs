#include "hip/hip_runtime.h"
/* 
   Copyright 2025 Mahdi Almusaad (https://github.com/mahdialmusaad)
   under the MIT License (https://opensource.org/license/mit)

   Using the Monte Carlo method, we can approximate pi by generating random 'points'
   on a 'unit square' (coordinates lie in range [0-1]) and using the ratio of the number
   of 'points' inside a 'quadrant' (i.e. a quarter-circle) on the square to the total 'points'
   generated, resulting in a value that converges on pi/4 as the number of 'points' approaches infinity.

   This specific Monte Carlo method is easily parallelized and easy to calculate as
   it only involves generating random points and calculating their length and an overall ratio,
   with no dependency on previous iterations.

   However, it is hopelessly inaccurate, only allowing calculations of a very few digits of pi even
   as the number of iterations is heavily increased.

   See https://en.wikipedia.org/wiki/Monte_Carlo_method and https://en.wikipedia.org/wiki/Pi#Monte_Carlo_methods for more information.

   This is the CUDA version of this program.
   The C version is available within the same repository (pi_mcarlo.c).
*/

/* Required includes. */
#include <string.h>
#include <stdint.h>
#include <stdio.h>


/*
   Function declarations.
*/


/* Error checking macro and function. */
#define checkError(result) { checkCudaError(result, __LINE__); }
__host__ void checkCudaError(hipError_t result, int line);

/* 
   Returns a 'pseudo-random' decimal in the range [0, 1] using the provided seed.
   Modifies the seed for the next call.

   This is marked with __device__ as it is only required for the below kernel function.
*/
__device__ float fastrand01(unsigned *seed);

/* 
   Generates a random 'point' for each thread on a unit square to determine
   if it is within a 'quadrant' for use in a Monte Carlo pi approximation.

   Takes in a pointer to some amount of bytes, depending on how many threads were given.

   After completion by a thread, either the first or second bit of each byte will be set. The first
   bit is changed if the point was outside otherwise the second if it was inside the 'quadrant'.

   The ratio of the number of 'inside points' and *total* 'points' converges on pi/4 as the total
   number of threads/number of stored results approaches infinity.

   This is marked with __global__ as it is a 'kernel' function (i.e. GPU thread code).
*/
__global__ void approximate_pi_mcarlo(uint8_t *results_array, time_t curr_time);

int main(int argc, char *argv[]) {
	/* Validate number of arguments. */
	if (argc != 3) {
		fprintf(stderr, "Usage: %s thread_blocks threads_per_block\n", *argv);
		return EXIT_FAILURE;
	}

	/* Get values from input. */
	const long num_thread_blocks = strtol(argv[1], NULL, 10);
	const long threads_per_block = strtol(argv[2], NULL, 10);
	const unsigned long total_thread_count = num_thread_blocks * threads_per_block;

	/* Validate given values. */
	if (num_thread_blocks <= 0 || threads_per_block <= 0) {
		fprintf(stderr, "Values must be positive.\n");
		return EXIT_FAILURE;
	}
	if (threads_per_block > 1024) {
		fprintf(stderr, "Too many threads per block.\n");
		return EXIT_FAILURE;
	}
	if (num_thread_blocks > INT32_MAX) {
		fprintf(stderr, "Too many thread blocks.\n");
		return EXIT_FAILURE;
	}

	printf("Using %lu threads\n", total_thread_count);

	/* Create an empty results array */
	uint_least64_t total_results[2];
	memset(&total_results, 0, sizeof total_results);
	
	/* Malloc a byte for each thread (using individual bits to store results) */
	uint8_t *thread_counters;
	const size_t results_size_bytes = sizeof *thread_counters * total_thread_count;
	checkError(hipMallocManaged(&thread_counters, results_size_bytes));

	/* Prefetch allocated memory to avoid page faults */
	checkError(hipMemPrefetchAsync(thread_counters, results_size_bytes, 0));

	/* Start calculating pi using explained method across a set amount of threads and blocks */
	approximate_pi_mcarlo<<<num_thread_blocks, threads_per_block>>>(thread_counters, time(NULL));

	/* Wait for calculations to complete (kernel code is non-blocking) */
	checkError(hipDeviceSynchronize());

	/* Combine all kernel results into the host array */
	for (size_t i = 0; i < total_thread_count; ++i) ++total_results[thread_counters[i] - 1];

	/* Free malloc'd counters */
	checkError(hipFree(thread_counters));

	/* Print overall counters results and pi from points ratio. */
	printf("Points results:\n  %lu inside\n  %lu outside\nPi approximation: %f\n",
		total_results[1], total_results[0],
		(4.0 * (double)total_results[1]) / (double)(total_results[0] + total_results[1])
	);

	return EXIT_SUCCESS;
}


/*
   Function definitions.
*/


__host__ void checkCudaError(hipError_t result, int line)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(result));
		exit(EXIT_FAILURE);
	}
}

__device__ float fastrand01(unsigned *seed)
{
	return (float)( (*seed = 3812762923u * (*seed)) & RAND_MAX) / (float)RAND_MAX;
}

__global__ void approximate_pi_mcarlo(uint8_t *results_array, time_t curr_time)
{
	/* Calculate this thread's global index. */
	const unsigned threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	/* Create a local seed value using the calculated index and given time for the 'random' function.
	   It is also possible to use cuRand (https://developer.nvidia.com/hiprand) here. */
	unsigned seed = ((~threadIndex * 23687u) ^ 965724u) * curr_time; 
	/* Get a random X and Y position, each in the range [0, 1]. */
	const float randX = fastrand01(&seed), randY = fastrand01(&seed);
	/* The point is 'inside' the 'circle' if the coordinate's square length (X^2 + Y^2) is less than 1.0.
	   Using the condition's result (0 or 1) to determine the bit to set. */
	results_array[threadIndex] |= 1 + (randX * randX + randY * randY < 1.0);
}
